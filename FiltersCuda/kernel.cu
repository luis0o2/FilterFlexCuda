#include "hip/hip_runtime.h"
﻿#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <iostream>
#include "stb_image.h"
#include "stb_image_write.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void rgbToGrayscaleKernel(unsigned char* d_in, unsigned char* d_out, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * 3;
        unsigned char r = d_in[idx];
        unsigned char g = d_in[idx + 1];
        unsigned char b = d_in[idx + 2];
        d_out[y * width + x] = 0.299f * r + 0.587f * g + 0.114f * b;
    }
}

void processImage(const char* inputPath, const char* outputPath) {
    int width, height, channels;
    unsigned char* img = stbi_load(inputPath, &width, &height, &channels, 0);
    if (!img) {
        std::cerr << "Error reading input image: " << inputPath << std::endl;
        return;
    }
    if (channels < 3) {
        std::cerr << "Image does not have 3 channels: " << inputPath << std::endl;
        stbi_image_free(img);
        return;
    }

    unsigned char* d_in, * d_out;
    size_t numPixels = width * height;
    size_t inputSize = numPixels * 3 * sizeof(unsigned char);
    size_t outputSize = numPixels * sizeof(unsigned char);

    hipError_t err;

    err = hipMalloc(&d_in, inputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        stbi_image_free(img);
        return;
    }

    err = hipMalloc(&d_out, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_in);
        stbi_image_free(img);
        return;
    }

    err = hipMemcpy(d_in, img, inputSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_in);
        hipFree(d_out);
        stbi_image_free(img);
        return;
    }

    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    rgbToGrayscaleKernel << <gridDim, blockDim >> > (d_in, d_out, width, height);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_in);
        hipFree(d_out);
        stbi_image_free(img);
        return;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_in);
        hipFree(d_out);
        stbi_image_free(img);
        return;
    }

    unsigned char* h_out = new unsigned char[outputSize];
    err = hipMemcpy(h_out, d_out, outputSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_in);
        hipFree(d_out);
        delete[] h_out;
        stbi_image_free(img);
        return;
    }

    stbi_write_png(outputPath, width, height, 1, h_out, width);

    std::cout << "Successfully wrote image to: " << outputPath << std::endl;

    hipFree(d_in);
    hipFree(d_out);
    delete[] h_out;
    stbi_image_free(img);
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <input_image_path> <output_image_path>" << std::endl;
        return 1;
    }

    const char* inputPath = argv[1];
    const char* outputPath = argv[2];

    processImage(inputPath, outputPath);
    return 0;
}
